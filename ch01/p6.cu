#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>

__global__ void helloFromGPU()
{
    // a thread index threadIdx.x is assigned to each thread in karnel
    if (threadIdx.x == 5)
    {
        printf("Hello World from GPU thread %d!\n", threadIdx.x);
    }
}

int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>();
    CHECK(hipDeviceSynchronize());
    return 0;
}
