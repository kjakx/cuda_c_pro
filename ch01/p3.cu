#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>();
    // device code will be executed
    CHECK(hipDeviceSynchronize());
    return 0;
}