#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>();
    CHECK(hipDeviceReset());
    return 0;
}

// compiled as follows:
// $ nvcc hello.cu -o hello
// gpu will work normally.