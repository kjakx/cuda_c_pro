#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>();
    // device code won't be executed
    // CHECK(hipDeviceReset());
    return 0;
}